#include "hip/hip_runtime.h"
#include "d.hpp"
#include <cstdio>

__global__ void kernel() {
  printf("kernel id = %d\n", blockIdx.x * blockDim.x + threadIdx.x);
}

int launch() {
  kernel<<<2, 3>>>();
  return 0;
}